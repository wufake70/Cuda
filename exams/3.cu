#include "hip/hip_runtime.h"
/*

3.利用GPU完成原地矩阵(3000*3000)转置操作:
要求:输入矩阵和输出矩阵在同一存储空间内, 不得另外申请空间.

*/

#include <stdio.h>
#include <stdlib.h>
#include "error.cuh"

#define TILE_DIM 32   //Don't ask me why I don't set these two values to one
#define BLOCK_SIZE 32
#define N 3000

__managed__ int input_M[N * N];      //input matrix & GPU result
int cpu_result[N * N];   //CPU result
//in-place matrix transpose

__global__ void ip_transpose(int* a) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < N&&i<j) {
        /*
        i<j,保证只有对角线下部分的线程能进行转换(也可以i>j),
        对角线上的线程对结果没影响，但影响整体速率，可直接优化掉
        1 2 3  第一列可进行交换的线程，对应的元素 4 7
        4 5 6  i=0,j=1
        7 8 9  i=0,j=2

        1 4 7
        2 5 8
        3 6 9
        */
        int tmp = a[i * N + j];
        a[i * N + j] = a[j * N + i];
        a[j * N + i] = tmp;
    }
}




void cpu_transpose(int* A, int* B)
{
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N; i++)
        {
            B[i * N + j] = A[j * N + i];
        }
    }
}

int main(int argc, char const* argv[])
{

    hipEvent_t start, stop_gpu;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop_gpu));


    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            input_M[i * N + j] = rand() % 1000;
        }
    }
    cpu_transpose(input_M, cpu_result);

    CHECK(hipEventRecord(start));
    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    ip_transpose <<<dimGrid, dimBlock >>>(input_M);
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));

    float elapsed_time_gpu;
    CHECK(hipEventElapsedTime(&elapsed_time_gpu, start, stop_gpu));
    printf("Time_GPU = %g ms.\n", elapsed_time_gpu);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop_gpu));

    int ok = 1;
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if (fabs(input_M[i * N + j] - cpu_result[i * N + j]) > (1.0e-10))
            {
                ok = 0;
            }
        }
    }


    if (ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }
   

    return 0;
}