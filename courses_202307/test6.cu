#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "error.cuh"

#define TILE_DIM 32   //Don't ask me why I don't set these two values to one
#define BLOCK_SIZE 32
#define N 3000

__managed__ int input_M[N * N];      //input matrix & GPU result
int cpu_result[N * N];   //CPU result

__global__ void ip_transpose(int* a, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n && i < j) {
        int tmp = a[i * n + j];
        a[i * n + j] = a[j * n + i];
        a[j * n + i] = tmp;
    }
}




void cpu_transpose(int* A, int* B)
{
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N; i++)
        {
            B[i * N + j] = A[j * N + i];
        }
    }
}

int main(int argc, char const* argv[])
{

    hipEvent_t start, stop_gpu;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop_gpu));


    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            input_M[i * N + j] = rand() % 1000;
        }
    }
    cpu_transpose(input_M, cpu_result);

    CHECK(hipEventRecord(start));
    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    ip_transpose <<<dimGrid, dimBlock >>>(input_M,N);
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));

    float elapsed_time_gpu;
    CHECK(hipEventElapsedTime(&elapsed_time_gpu, start, stop_gpu));
    printf("Time_GPU = %g ms.\n", elapsed_time_gpu);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop_gpu));

    int ok = 1;
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if (fabs(input_M[i * N + j] - cpu_result[i * N + j]) > (1.0e-10))
            {
                ok = 0;
            }
        }
    }


    if (ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }
    for(int i=0;i<100;i++) printf("%d %s",cpu_result[i],(i+1)%10==0?"\n":"");

    printf("\n");
    for(int i=0;i<100;i++) printf("%d %s",input_M[i],(i+1)%10==0?"\n":"");

    return 0;
}